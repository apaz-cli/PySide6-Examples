#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// CUDA kernel for matrix multiplication
__global__ void matrixMul(const float* a, const float* b, float* c, 
                         int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

// Shared memory optimized matrix multiplication
__global__ void matrixMulShared(const float* a, const float* b, float* c,
                               int m, int n, int k) {
    __shared__ float tile_a[16][16];
    __shared__ float tile_b[16][16];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float sum = 0.0f;
    
    for (int tile = 0; tile < (k + 15) / 16; ++tile) {
        // Load tiles into shared memory
        if (row < m && tile * 16 + threadIdx.x < k) {
            tile_a[threadIdx.y][threadIdx.x] = a[row * k + tile * 16 + threadIdx.x];
        } else {
            tile_a[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        if (col < n && tile * 16 + threadIdx.y < k) {
            tile_b[threadIdx.y][threadIdx.x] = b[(tile * 16 + threadIdx.y) * n + col];
        } else {
            tile_b[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial sum
        for (int i = 0; i < 16; ++i) {
            sum += tile_a[threadIdx.y][i] * tile_b[i][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    if (row < m && col < n) {
        c[row * n + col] = sum;
    }
}

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    } while(0)

int main() {
    const int n = 1024 * 1024;  // 1M elements
    const int size = n * sizeof(float);
    
    // Host vectors
    std::vector<float> h_a(n), h_b(n), h_c(n);
    
    // Initialize host vectors
    for (int i = 0; i < n; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }
    
    // Device vectors
    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(&d_a, size));
    CUDA_CHECK(hipMalloc(&d_b, size));
    CUDA_CHECK(hipMalloc(&d_c, size));
    
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b.data(), size, hipMemcpyHostToDevice));
    
    // Launch configuration
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    
    // Time the kernel execution
    auto start = std::chrono::high_resolution_clock::now();
    
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    CUDA_CHECK(hipDeviceSynchronize());
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_c.data(), d_c, size, hipMemcpyDeviceToHost));
    
    // Verify result
    bool success = true;
    for (int i = 0; i < std::min(10, n); ++i) {
        float expected = h_a[i] + h_b[i];
        if (std::abs(h_c[i] - expected) > 1e-5) {
            success = false;
            break;
        }
    }
    
    std::cout << "Vector addition " << (success ? "PASSED" : "FAILED") << std::endl;
    std::cout << "Execution time: " << duration.count() << " microseconds" << std::endl;
    
    // Get device properties
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    std::cout << "Device: " << prop.name << std::endl;
    std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Shared memory per block: " << prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
    
    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    
    return 0;
}
